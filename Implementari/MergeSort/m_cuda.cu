#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N1000 100000000


__device__ void CudaMerge(int* values, int* results, int l, int r, int u)
{
	int i, j, k;
	i = l; j = r; k = l;

	while (i < r && j < u) {
		if (values[i] <= values[j]) { results[k] = values[i]; i++; }
		else { results[k] = values[j]; j++; }
		k++;
	}

	while (i < r) {
		results[k] = values[i]; i++; k++;
	}

	while (j < u) {
		results[k] = values[j]; j++; k++;
	}
	for (k = l; k < u; k++) {
		values[k] = results[k];
	}
}


__global__ static void CudaMergeSort(int * values, int* results, int dim){
	extern __shared__ int shared[];


	const unsigned int tid = threadIdx.x;
	int k, u, i;
	shared[tid] = values[tid];
	
	__syncthreads();
	k = 1;
	while (k <= dim)
	{
		i = 0;
		while (i + k < dim)
		{
			u = i + k * 2;;
			if (u > dim)
			{
				u = dim + 1;
			}
			CudaMerge(shared, results, i, i + k, u);
			i = i + k * 2;
		}
		k = k * 2;

		__syncthreads();
	}

	values[tid] = shared[tid];
}


void mergeSort(int arr[], int p, int q);


int main(int argc, char** argv)
{	
	float elapsed1000 = 0;

	hipEvent_t start1000, stop1000;
	
	
	hipEventCreate(&start1000);
	hipEventCreate(&stop1000);
	
	int  *values4;
	int  *values4s;
	 
	values4 = (int*)malloc(N1000*sizeof(int));
	values4s = (int*)malloc(N1000*sizeof(int));
	
	int* dvalues, *results;
	
	hipMalloc((void**)&dvalues, sizeof(int) * N1000);
	hipMalloc((void**)&results, sizeof(int) * N1000);

	printf("\nElements for N = 1000:\n");
	for (int i = 0; i < N1000; i++)
	{
		values4[i] = rand();
		values4s[i] = values4[i];
		//printf("%d ", values4[i]);
	}

	printf("\n");
	//Start monitorizing of cuda operations
	hipEventRecord(start1000, 0);
	//Generation cuda variables ables to work and copying the variables from host to device
	
	hipMemcpy(dvalues, values4, sizeof(int) * N1000, hipMemcpyHostToDevice);
	
	hipMemcpy(results, values4s, sizeof(int)* N1000, hipMemcpyHostToDevice);
	//Calling algorithm MergeSort
	
	CudaMergeSort << <1, N1000, sizeof(int) * N1000 * 2 >> > (dvalues, results, N1000);

	// Freeing memory space used and returning values sortered
	hipFree(dvalues);
	hipMemcpy(values4, results, sizeof(int)*N1000, hipMemcpyDeviceToHost);
	hipFree(results);
	//Stopping time monitoring
	hipEventRecord(stop1000, 0);
	hipEventSynchronize(stop1000);
	//Calculating the total time of execution
	hipEventElapsedTime(&elapsed1000, start1000, stop1000);
	// Freeing the events created before
	hipEventDestroy(start1000);
	hipEventDestroy(stop1000);
	//Showing sorted elements
	
	//Showing the time of execution
	printf("\n\t||| The elapsed time in gpu was %.2f ms |||", elapsed1000);
	printf("\n");

	hipDeviceReset();
	hipDeviceReset();

	

	return 0;
}


void merge(int arr[], int p, int q, int r) {

	int i, j, k;
	int n1 = q - p + 1;
	int n2 = r - q;
	
	//arrs temporales
	int *L, *M;
	hipMalloc((void**)&L, sizeof(int) * n1);
	hipMalloc((void**)&M, sizeof(int) * n2);
	hipMalloc((void**)&arr, sizeof(int) * N1000);
	

	for (int i = 0; i < n1; i++)
	{
		L[i] = arr[p + i];
	}

	for (int j = 0; j < n2; j++)
	{
		M[j] = arr[q + 1 + j];
	}

	i = 0;
	j = 0;
	k = p;

	while (i < n1 && j < n2)
	{
		if (L[i] <= M[j])
		{
			arr[k] = L[i];
			i++;
		}
		else
		{
			arr[k] = M[j];
			j++;
		}
		k++;
	}

	while (i < n1)
	{
		arr[k] = L[i];
		i++;
		k++;
	}

	while (j < n2)
	{
		arr[k] = M[j];
		j++;
		k++;
	}
}

void mergeSort(int arr[], int p, int q) {


	if (p < q)
	{
		int mitad = (p + q) / 2;

		mergeSort(arr, p, mitad);
		mergeSort(arr, mitad + 1, q);
		merge(arr, p, mitad, q);
	}
}
